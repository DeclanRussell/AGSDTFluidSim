#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------------------------------------------
/// @file CudaSPHKernals.cu
/// @author Declan Russell
/// @date 08/03/2015
/// @version 1.0
//----------------------------------------------------------------------------------------------------------------------
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include "CudaSPHKernals.h"
#include "hip/hip_vector_types.h"  //< some math operations with cuda types

#define pi 3.14159265359f

//----------------------------------------------------------------------------------------------------------------------
/// @brief Kernal designed to produce a has key based on the location of a particle
/// @brief Hash function taken from Teschner, M., Heidelberger, B., Mueller, M., Pomeranets, D. and Gross, M.
/// @brief (2003). Optimized spatial hashing for collision detection of deformable objects
/// @param d_hashArray - pointer to a buffer to output our hash keys
/// @param d_posArray - pointer to the buffer that holds our particle positions
/// @param numParticles - the number of particles in our buffer
/// @param resolution - the resolution of our hash table
/// @param _gridScaler - Scales our points to between 0-1.
__global__ void pointHash(unsigned int* d_hashArray, float3* d_posArray, unsigned int numParticles, float resolution, float _gridScaler){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //dont want to start accessing data that doesn't exist! Could be deadly!
    if(idx<numParticles){
        //calculate our hash key and store it in our hash key array
        float3 normalizeCoords = d_posArray[idx]*_gridScaler;
        //if our normalized coords are not between 0-1 then we need to
        if(normalizeCoords.x<0){
            normalizeCoords.x = 0;
        }
        if(normalizeCoords.y<0){
            normalizeCoords.y = 0;
        }
        if(normalizeCoords.z<0){
            normalizeCoords.z = 0;
        }
        if(normalizeCoords.x>1){
            normalizeCoords.x = 1;
        }
        if(normalizeCoords.y>1){
            normalizeCoords.y = 1;
        }
        if(normalizeCoords.z>1){
            normalizeCoords.z = 1;
        }

        float3 gridPos = floorf(normalizeCoords*resolution);

        //give our particles a hash value
        d_hashArray[idx] = gridPos.x * resolution * resolution + gridPos.y * resolution + gridPos.z;
    }
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This kernal is designed to count the cell occpancy of a hash table
/// @param d_hashArray - pointer to hash table buffer
/// @param d_cellOccArray - output array of cell occupancy count
/// @param _hashTableSize - the size of our hash table
/// @param _numPoints - the number of particles in our hashed array
__global__ void countCellOccKernal(unsigned int *d_hashArray, unsigned int *d_cellOccArray, int _hashTableSize, unsigned int _numPoints){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure our idx is valid and add the occupancy count to the relevant cell
    if ((idx < _numPoints) && (d_hashArray[idx] < _hashTableSize)) {
        atomicAdd(&(d_cellOccArray[d_hashArray[idx]]), 1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our desity weighting kernal used in our navier stokes equations
/// @param _dst - the distance away of the neighbouring
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _densKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float densityWeighting(float _dst,float _smoothingLength, float _densKernConst){
    float weighting = 0;
    if(_dst>0 && _dst<=_smoothingLength){
        float temp = (_smoothingLength * _smoothingLength) - (_dst*_dst);
        weighting = _densKernConst * temp * temp * temp;
    }
    return weighting;
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our desity weighting kernal used in our navier stokes equations
/// @param _r - vector from our neighbour particle to our current particle
/// @param _dst - the distance between our particles
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _pressKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float3 pressureWeighting(float3 _r, float _dst,float _smoothingLength, float _pressKernConst){
    float weighting = 0.f;
    if(_dst>0 && _dst<=_smoothingLength){
        weighting = _pressKernConst * (_smoothingLength-_dst) * (_smoothingLength-_dst);
        _r/=_dst;
    }
    _r *= weighting;
    return _r;
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our viscosty weighting kernal used in our navier stokes equations
/// @param _r - vector from our neighbour particle to our current particle
/// @param _dst - the distance between our particles
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _viscKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float3 viscosityWeighting(float3 _r, float _dst,float _smoothingLength, float _viscKernConst){
    float weighting = 0;
    if(_dst>0 && _dst<=_smoothingLength){
        weighting = _viscKernConst * _smoothingLength - _dst;
    }
    _r *= weighting;
    return _r;
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void fluidSolverPerCellKernal(int _maxSamples, float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int *d_cellOccArray, unsigned int *d_cellIndxArray, int _hashResolution, int _hashTableSize, float _smoothingLength, float _timestep, float _particleMass, float _restDensity, float _gasConstant, float _visCoef, float densKernConst, float pressKernConst, float viscKernConst){


    // In this solver we will be exploiting the shared memory of the this block
    // to store our neighbouring particles properties rather than loading it
    // from a buffer.
    // This gives us great speed advantages! So hold on to your seats!
    // Firstly lets declare our shared piece of memory
    __shared__ int cellOcc;
    __shared__ unsigned short int sampleSum;
    __shared__ unsigned int cellStartIdx;

    if(threadIdx.x==0){
        sampleSum=0;
        // Read in our how many particles our cell holds
        cellOcc = d_cellOccArray[blockIdx.x];
        // Calculate our index for these particles in our buffer
        cellStartIdx = d_cellIndxArray[blockIdx.x];
    }
    __syncthreads();
    //If there is nothing in the cell its faster if we dont declare this shared
    //memory as just allocating it takes time!
    if(!cellOcc) return;
    extern __shared__ particleCellProp nParticleData[];
    sampleSum = min(_maxSamples,cellOcc);

    int i;
    if(threadIdx.x==0){
        //load in our neighbouring cells
        int resX = _hashResolution * _hashResolution;
        unsigned int x;
        unsigned int y;
        unsigned int nCellIdx;
        int j;
        for(i=0;i<27;i++){
            if(sampleSum>=_maxSamples) break;
            x = floor((float)i/(float)9);
            y = floor((float)(i -9*x)/(float)3);
            nCellIdx = blockIdx.x +  (x-1u) * resX + (y-1u) * _hashResolution + (i%3-1);
            if(nCellIdx>0&&nCellIdx<_hashTableSize){
                if(nCellIdx==blockIdx.x) continue;
                int nCellStart = d_cellIndxArray[nCellIdx];
                //see how much space we have left in our shared memory
                int nStart = sampleSum;
                int dif = min(_maxSamples-nStart,d_cellOccArray[nCellIdx]);
                sampleSum+=dif;
                //load in our neightbour data
                for(j=0; j<dif;j++){
                    nParticleData[nStart+j].idx = nCellStart+j;
                }
            }
        }
    }
    __syncthreads();
    float3 curPartPos;
    float3 curPartVel;
    int curPartIdx;
    if(threadIdx.x<sampleSum){
        if(threadIdx.x<cellOcc){
            curPartIdx = cellStartIdx+threadIdx.x;
            curPartPos = nParticleData[threadIdx.x].pos = d_posArray[curPartIdx];
            curPartVel = nParticleData[threadIdx.x].vel = d_velArray[curPartIdx];
        }
        else{
            curPartPos = nParticleData[threadIdx.x].pos = d_posArray[nParticleData[threadIdx.x].idx];
            curPartVel = nParticleData[threadIdx.x].vel = d_velArray[nParticleData[threadIdx.x].idx];
        }
    }

    // Calculate the density of our particle
    float density = 0.0;
    for(i=0;i<sampleSum; i++){
        density += _particleMass * densityWeighting(length(curPartPos-nParticleData[i].pos),_smoothingLength,densKernConst);
    }
    //load result into our shared memory
    if(threadIdx.x<sampleSum){
        nParticleData[threadIdx.x].density = density;
    }

    //make sure that what were going to access is in our range
    if(threadIdx.x<cellOcc){
        //multiplication faster than divide so lets just do it once
        float invDensity = 1.0f/density;
        __syncthreads();
        //Once this is done we can finally do some navier-stokes!!
        float3 viscosityForce = make_float3(0.0f,0.0f,0.0f);
        float3 tensionSum = make_float3(0);
        float3 tensionSumTwo = make_float3(0);
        float3 acc = make_float3(0.0f,0.0f,0.0f);
        {
            float massDivDen;
            float3 pressWeightTemp, viscWeightTemp, r;
            float currPressTemp,nPressTemp,p1,p2,dst;
            //calculate the pressure force of our current particle
            currPressTemp = (_gasConstant * (density - _restDensity));
            p1 = (currPressTemp*invDensity*invDensity);
            for(i=0;i<sampleSum;i++){
                //cuda seems to struggle when testing for >0 No idea why
                if(nParticleData[i].density>1 && density>1){
                //if(nParticleData[i].density>0 && density>0){
                    //calculate the pressure force of our neighbour particle
                    nPressTemp = (_gasConstant * (nParticleData[i].density - _restDensity));
                    p2 = (nPressTemp/(nParticleData[i].density*nParticleData[i].density));
                    r = curPartPos - nParticleData[i].pos;
                    dst = length(r);
                    pressWeightTemp = pressureWeighting(r,dst,_smoothingLength,pressKernConst);
                    acc -= ( p1 + p2 ) * pressWeightTemp;
                    //if(acc.x!=acc.x){
                    //    printf("balls acc %f,%f,%f cp %f cd %f cid %f np %f \nnd %f m %f pw %f,%f,%f cTn %f,%f,%f\n",acc.x,acc.y,acc.z,currPressTemp,density,invDensity,nPressTemp,nParticleData[i].density,_restDensity,_particleMass,pressWeightTemp.x,pressWeightTemp.y,pressWeightTemp.z,r.x,r.y,r.z);
                    //    if(density>0) printf("cd %f\n",density);
                    //}
                    //if(length(acc)>12)printf("acc %f,%f,%f  den %f nDen %f _dst %f w%f,%f,%f   cellOcc %d  nIdex %d\n",acc.x,acc.y,acc.z,density,nParticleData[i].density,dst,pressWeightTemp.x,pressWeightTemp.y,pressWeightTemp.z,cellOcc,i);
                    //calculate our viscosity force
                    viscWeightTemp = viscosityWeighting(r,dst,_smoothingLength,viscKernConst);
                    massDivDen = _particleMass/nParticleData[i].density;
                    viscosityForce += (curPartVel - nParticleData[i].vel) * massDivDen * viscWeightTemp;
                    //this is needed for surface tension
                    tensionSum += massDivDen * pressWeightTemp;
                    tensionSumTwo += massDivDen * viscWeightTemp;
                }
            }
        }


//        //calculate our surface tension
//        float3 tensionForce = make_float3(0);
//        {
//            float nLength = length(tensionSumTwo);
//            //1.0 is currently our threshold as tension becomes very unstable as n approaches 0
//            if(nLength>0.5){
//                //this also needs to be multipied by our tension contant
//                tensionForce = (tensionSumTwo/nLength) * tensionSum;
//            }
//        }

        acc*=_particleMass;
        acc+=make_float3(0.0f,-9.8f,0.0f);
        //if(density>0) acc+= (viscosityForce*_visCoef /*+ tensionForce*/)*invDensity;


        //calculate our new velocity
        //euler intergration (Rubbish over large time steps)
//        float3 newVel = curPartVel + (acc * _timestep);
//        float3 newPos = curPartPos + (newVel * _timestep);

        //leap frog integration
        //more stable if we move by half steps than full
        float3 velHalfBack = curPartVel - 0.5f * _timestep * d_accArray[curPartIdx];
        float3 velHalfFor = velHalfBack + _timestep * acc;
        //printf("pf %f,%f,%f vel %f,%f,%f\n",acc.x,acc.y,acc.z,velHalfFor.x,velHalfFor.y,velHalfFor.z);


        //XSPH velocity correction
        //can be found in Paiva, A., Petronetto, F., Lewiner, T. and Tavares, G. (2009).
        //Particle-based viscoplastic fluid/solid simulation,
        //To achieve this lets take advantage of our shared memory again
        nParticleData[threadIdx.x].vel = velHalfFor;
        __syncthreads();
        float3 newVel = make_float3(.0f);
        for(i=0;i<cellOcc;i++){
            if((density>0)&&(nParticleData[i].density>0))
            newVel += (2.0f*_particleMass/(density+nParticleData[i].density)) * (nParticleData[i].vel - velHalfBack) * densityWeighting(length(curPartPos-nParticleData[i].pos),_smoothingLength,densKernConst);
        }

        newVel = velHalfFor + 0.3f * newVel;
        float3 newPos = curPartPos + (newVel * _timestep);
        //printf("p %f,%f,%f\n",newPos.x,newPos.y,newPos.z);
//        if(newPos.x!=newPos.x){
//            printf("shit currentpos %f,%f,%f newPos %f,%f,%f samples %d\n",curPartPos.x,curPartPos.y,curPartPos.z,newPos.x,newPos.y,newPos.z,sampleSum);
//            printf("vel: %f,%f,%f\n",newVel.x,newVel.y,newVel.z);
//            printf("timeStep %f acc %f,%f,%f\n\n",_timestep,acc.x,acc.y,acc.z);
//        }
        //update our particle positin and velocity
        d_velArray[curPartIdx] = newVel;
        d_posArray[curPartIdx] = newPos;
        d_accArray[curPartIdx] = acc;


    }
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void collisionDetKernal(planeProp *d_planeArray, unsigned int _numPlanes, float3 *d_posArray, float3 *d_velArray, unsigned int _numParticles, float _timeStep){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //Make sure we're in our range
    if(idx<_numParticles){
        // Exploit some shared memory for fast access of our plane information
        extern __shared__ planeProp planes[];
        if(threadIdx.x<_numPlanes){
            planes[threadIdx.x] = d_planeArray[threadIdx.x];
        }


        // start and end points of our line segement
        float3 vel = d_velArray[idx];
        float3 newVel = vel;
        float3 pos = d_posArray[idx];
        float dst = 0;
        bool intersect = false;
        float3 pToO;
        int itersectPlane;
        //iterate through planes
        for(int i=0; i<_numPlanes; i++){
            //if its on the wrong side of the plane move it back and reflect the velocity
            //this is not 100% accurate collision, but at least it works!!
            if(dot(pos-planes[i].pos,planes[i].normal)<0.f){
                //move particle to closest point on plane
                pToO = pos-planes[i].pos;
                dst = dot(pToO,planes[i].normal);
                pos = pos-(dst*planes[i].normal);
                //reflect velocity arount normal
                newVel =  newVel - (2.0f * dot(newVel,planes[i].normal) * planes[i].normal);
                itersectPlane = i;
                intersect=true;
            }
        }

        //if intersect has occured move our particle back and change our velocity
        if(intersect==true){
            d_posArray[idx] = pos;
            d_velArray[idx] = newVel - (1.0 - planes[itersectPlane].restCoef) * newVel * fabs(planes[itersectPlane].normal);
        }
    }
}

//----------------------------------------------------------------------------------------------------------------------
void createHashTable(unsigned int* d_hashArray, float3* d_posArray, unsigned int _numParticles, float _smoothingLength, float _gridSize, int _maxNumThreads){
    //std::cout<<"createHashTable"<<std::endl;
    //calculate how many blocks we want
    int blocks = ceil(_numParticles/_maxNumThreads)+1;
    pointHash<<<_numParticles,1>>>(d_hashArray,d_posArray,_numParticles,_gridSize/_smoothingLength,1.0f/_gridSize);


    //DEBUG: uncomment to print out counted cell occupancy, WARNING SUPER SLOW!
    //thrust::device_ptr<unsigned int> t_hashPtr = thrust::device_pointer_cast(d_hashArray);
    //thrust::copy(t_hashPtr, t_hashPtr+_numParticles, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<"\n"<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("createHashTable CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void sortByKey(unsigned int *d_hashArray, float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int _numParticles){
    //std::cout<<"sortByKey"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //thrusts sort algorithm
    thrust::device_ptr<unsigned int> t_hashPtr = thrust::device_pointer_cast(d_hashArray);
    thrust::device_ptr<float3> t_posPtr = thrust::device_pointer_cast(d_posArray);
    thrust::device_ptr<float3> t_velPtr = thrust::device_pointer_cast(d_velArray);
    thrust::device_ptr<float3> t_accPtr = thrust::device_pointer_cast(d_accArray);


    //sort our buffers
    thrust::sort_by_key(t_hashPtr,t_hashPtr+_numParticles, thrust::make_zip_iterator(thrust::make_tuple(t_posPtr,t_velPtr,t_accPtr)));


    //DEBUG: uncomment to print out sorted hash keys
    //thrust::copy(t_hashPtr, t_hashPtr+_numParticles, std::ostream_iterator<unsigned int>(std::cout, " "));
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("sortByKey CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}
//----------------------------------------------------------------------------------------------------------------------
void countCellOccupancy(unsigned int *d_hashArray, unsigned int *d_cellOccArray,unsigned int _hashTableSize, unsigned int _numPoints, unsigned int _maxNumThreads){
    //std::cout<<"countCellOccupancy"<<std::endl;
    //calculate how many blocks we want
    int blocks = ceil(_numPoints/_maxNumThreads)+1;
    countCellOccKernal<<<blocks,_maxNumThreads>>>(d_hashArray,d_cellOccArray,_hashTableSize,_numPoints);


    //DEBUG: uncomment to print out counted cell occupancy
    //thrust::device_ptr<unsigned int> t_occPtr = thrust::device_pointer_cast(d_cellOccArray);
    //thrust::copy(t_occPtr, t_occPtr+_hashTableSize, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<"\n"<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("countCellOccupancy CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void fillUint(unsigned int *_pointer, unsigned int _arraySize, unsigned int _fill){
    //std::cout<<"fillUint"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //them in thrust fill
    thrust::device_ptr<unsigned int> t_Ptr = thrust::device_pointer_cast(_pointer);
    //fill our buffer
    thrust::fill(t_Ptr, t_Ptr+_arraySize, _fill);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("FillUint CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}
//----------------------------------------------------------------------------------------------------------------------
void createCellIdx(unsigned int* d_cellOccArray, unsigned int _size,unsigned int* d_cellIdxArray){
    //std::cout<<"createCellIdx"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //them in thrust
    thrust::device_ptr<unsigned int> t_cellOccPtr = thrust::device_pointer_cast(d_cellOccArray);
    thrust::device_ptr<unsigned int> t_cellIdxPtr = thrust::device_pointer_cast(d_cellIdxArray);
    //run an excludive scan on our arrays
    thrust::exclusive_scan(t_cellOccPtr,t_cellOccPtr+_size,t_cellIdxPtr);

    //DEBUG: uncomment to print out cell index buffer
    //thrust::copy(t_cellIdxPtr, t_cellIdxPtr+_size, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("createCellIdx CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void fluidSolver(float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int *d_cellOccArray, unsigned int *d_cellIndxArray, unsigned int _hashTableSize, int _hashResolution, unsigned int _maxNumThreads, float _smoothingLength, float _timestep, float _particleMass, float _restDensity, float _gasConstant, float _visCoef, float _densKernConst, float _pressKernConst, float _viscKernConst){
    //std::cout<<"fluidSolver"<<std::endl;
    //printf("memory allocated: %d",_maxNumThreads*(sizeof(particleCellProp)));
    //fluidSolverKernal<<<_hashTableSize, 30>>>(d_posArray,d_velArray,d_cellOccArray,d_cellIndxArray,_smoothingLength,_timestep, _particleMass, _restDensity,_gasConstant,_visCoef, _densKernConst, _pressKernConst, _viscKernConst);

    int totalSamples = 300;
    fluidSolverPerCellKernal<<<_hashTableSize,totalSamples,totalSamples*sizeof(particleCellProp)>>>(totalSamples,d_posArray,d_velArray,d_accArray,d_cellOccArray,d_cellIndxArray,_hashResolution,_hashTableSize,_smoothingLength,_timestep,_particleMass,_restDensity,_gasConstant,_visCoef,_densKernConst,_pressKernConst,_viscKernConst);


    //dynamic parrallelism version. Slow!
//    int blocks = ceil(10000/_maxNumThreads)+1;
//    fluidSolverKernalDP<<<blocks,_maxNumThreads>>>(d_posArray,d_velArray,d_accArray,d_cellOccArray,d_cellIndxArray,_hashResolution,_hashTableSize,_smoothingLength,_timestep,_particleMass,_restDensity,_gasConstant,_visCoef,_densKernConst,_pressKernConst,_viscKernConst);

    //std::cout<<std::endl;

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("Fluid solver CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}

//----------------------------------------------------------------------------------------------------------------------
void collisionDetectionSolver(planeProp *d_planeArray, unsigned int _numPlanes, float3 *d_posArray, float3 *d_velArray, float _timeStep, unsigned int _numParticles, unsigned int _maxNumThreads){
    //calculate how many blocks we want
    int blocks = ceil(_numParticles/_maxNumThreads)+1;
    //launch collision solver
    collisionDetKernal<<<blocks,_maxNumThreads,_numPlanes*sizeof(planeProp)>>>(d_planeArray,_numPlanes,d_posArray,d_velArray,_numParticles,_timeStep);

}
//----------------------------------------------------------------------------------------------------------------------

#include "hip/hip_runtime.h"
//----------------------------------------------------------------------------------------------------------------------
/// @file CudaSPHKernals.cu
/// @author Declan Russell
/// @date 08/03/2015
/// @version 1.0
//----------------------------------------------------------------------------------------------------------------------
#include <math.h>
#include <thrust/sort.h>
#include <thrust/fill.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include "CudaSPHKernals.h"
#include "cutil_math.h"  //< some math operations with cuda types

#define pi 3.14159265359f

//----------------------------------------------------------------------------------------------------------------------
/// @brief Kernal designed to produce a has key based on the location of a particle
/// @brief Hash function taken from Teschner, M., Heidelberger, B., Mueller, M., Pomeranets, D. and Gross, M.
/// @brief (2003). Optimized spatial hashing for collision detection of deformable objects
/// @param d_hashArray - pointer to a buffer to output our hash keys
/// @param d_posArray - pointer to the buffer that holds our particle positions
/// @param numParticles - the number of particles in our buffer
/// @param resolution - the resolution of our hash table
/// @param _gridScaler - Scales our points to between 0-1.
__global__ void pointHash(unsigned int* d_hashArray, float3* d_posArray, unsigned int numParticles, float resolution, float _gridScaler){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //dont want to start accessing data that doesn't exist! Could be deadly!
    if(idx<numParticles){
        //calculate our hash key and store it in our hash key array
        float3 normalizeCoords = d_posArray[idx]*_gridScaler;
        //if our normalized coords are not between 0-1 then we need to
        if(normalizeCoords.x<0){
            normalizeCoords.x = 0;
        }
        if(normalizeCoords.y<0){
            normalizeCoords.y = 0;
        }
        if(normalizeCoords.z<0){
            normalizeCoords.z = 0;
        }
        if(normalizeCoords.x>1){
            normalizeCoords.x = 1;
        }
        if(normalizeCoords.y>1){
            normalizeCoords.y = 1;
        }
        if(normalizeCoords.z>1){
            normalizeCoords.z = 1;
        }

        float3 gridPos = floor(normalizeCoords*resolution);

        //give our particles a hash value
        d_hashArray[idx] = gridPos.x * resolution * resolution + gridPos.y * resolution + gridPos.z;
    }
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This kernal is designed to count the cell occpancy of a hash table
/// @param d_hashArray - pointer to hash table buffer
/// @param d_cellOccArray - output array of cell occupancy count
/// @param _hashTableSize - the size of our hash table
/// @param _numPoints - the number of particles in our hashed array
__global__ void countCellOccKernal(unsigned int *d_hashArray, unsigned int *d_cellOccArray, int _hashTableSize, unsigned int _numPoints){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Make sure our idx is valid and add the occupancy count to the relevant cell
    if ((idx < _numPoints) && (d_hashArray[idx] < _hashTableSize)) {
        atomicAdd(&(d_cellOccArray[d_hashArray[idx]]), 1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our desity weighting kernal used in our navier stokes equations
/// @param _currentPos - the postions of the particle we are solving for
/// @param _neighPos - the position of the neighbouring particle we wish to calculate the weighting for
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _densKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float densityWeighting(float3 _currentPos, float3 _neighPos,float _smoothingLength, float _densKernConst){
    float rLength = length(_currentPos - _neighPos);
    float weighting = 0;
    if(rLength>0 && rLength<=_smoothingLength){
        float dst = (_smoothingLength * _smoothingLength) - (rLength*rLength);
        weighting = _densKernConst * dst * dst * dst;
    }
    return weighting;
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our desity weighting kernal used in our navier stokes equations
/// @param _currentPos - the postions of the particle we are solving for
/// @param _neighPos - the position of the neighbouring particle we wish to calculate the weighting for
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _pressKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float3 pressureWeighting(float3 _currentPos, float3 _neighPos,float _smoothingLength, float _pressKernConst){
    float3 r = _currentPos - _neighPos;
    float rLength = length(r);
    float weighting = 0;
    if(rLength>0 && rLength<=_smoothingLength){
        weighting = _pressKernConst * (_smoothingLength-rLength) * (_smoothingLength-rLength);
        r /= rLength;
    }
    r *= weighting;
    return r;
}
//----------------------------------------------------------------------------------------------------------------------
/// @brief This is our viscosty weighting kernal used in our navier stokes equations
/// @param _currentPos - the postions of the particle we are solving for
/// @param _neighPos - the position of the neighbouring particle we wish to calculate the weighting for
/// @param _smoothingLength - the smoothing length of our simulation. Can be thought of a hash cell size.
/// @param _viscKernCosnt - constant part of our kernal. Easier to calculate once on CPU and have loaded into device kernal.
/// @return return the weighting that our neighbouring particle has on our current particle
__device__ float3 viscosityWeighting(float3 _currentPos, float3 _neighPos,float _smoothingLength, float _viscKernConst){
    float3 r = _currentPos - _neighPos;
    float rLength = length(r);
    float weighting = 0;
    if(rLength>0 && rLength<=_smoothingLength){
        weighting = _viscKernConst * _smoothingLength - rLength;
    }
    r *= weighting;
    return r;
}

//----------------------------------------------------------------------------------------------------------------------
__global__ void fluidSolverPerCellKernal(float3 *d_posArray, float3 *d_velArray, float3 *d_accArray,unsigned int _particleIdx, unsigned int _cellOcc, float _smoothingLength, float _timestep, float _particleMass, float _restDensity, float _gasConstant, float _visCoef, float densKernConst, float pressKernConst, float viscKernConst){

    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //make sure that what were going to access is in our range
    if(idx<_cellOcc){
        int particleIdx = _particleIdx + idx;
        // In this solver we will be exploiting the shared memory of the this block
        // to store our neighbouring particles properties rather than loading it
        // from a buffer.
        // This gives us great speed advantages! So hold on to your seats!
        // Firstly lets declare our shared piece of memory
        __shared__ particleProp nParticleData[70];

        //lets load in our particles properties to our peice of shared memory
        //Due to limits on threads if we have more particles to this key than
        //Threads we may have to sacrifice some particles to sample for less
        //overhead but hopefully we can keep this under control by having a
        //good cell size (smoothing length) in our hash function.
        //While we're at it lets store our current particle position.
        float3 curPartPos = d_posArray[particleIdx];
        float3 curPartVel = d_velArray[particleIdx];
        int samples = min(_cellOcc,70);
        if(threadIdx.x<samples){
            nParticleData[threadIdx.x].pos = curPartPos;
            nParticleData[threadIdx.x].vel = curPartVel;
        }
        //sync our threads to make sure all our particle info has been copied
        //to shared memory
        __syncthreads();

        // Calculate the density of our particle
        // Possibly could optimise this with more dynamic parralism.
        // However for such a small loop this may not actually make
        // much difference. We also will have to take into count
        // kernal launch times & memory access.
        float density = 0.0;
        float3 nPartPosTemp;
        int i;
        for(i=0;i<samples; i++){
            nPartPosTemp = nParticleData[i].pos;
            density += _particleMass * densityWeighting(curPartPos,nPartPosTemp,_smoothingLength,densKernConst);
        }
        if(threadIdx.x<samples){
            nParticleData[threadIdx.x].density = density;
        }

        //Once this is done we can finally do some navier-stokes!!
        float3 pressureForce = make_float3(0,0,0);
        float3 viscosityForce = make_float3(0.0f,0.0f,0.0f);
        float3 pressWeightTemp, viscWeightTemp;
        float3 tensionSum = make_float3(0);
        float3 tensionSumTwo = make_float3(0);

        float nPartDenTemp;
        float massDivDen;
        float currPressTemp,nPressTemp,p1,p2,p1a;
        for(i=0;i<samples;i++){
            if(threadIdx.x == i) continue;
            nPartPosTemp = nParticleData[i].pos;
            nPartDenTemp = nParticleData[i].density;


            //calculate the pressure force
            currPressTemp = (_gasConstant * (density - _restDensity));
            p1 = (1.0f/currPressTemp);
            //printf("p1 %f den %f rest %f gs %f\n",p1,density,_restDensity,_gasConstant);
//            printf("den %f rest %f gs %f",density,_restDensity,_gasConstant);
//            p1a = (1.0f/currPressTemp);
            nPressTemp = (_gasConstant * (nPartDenTemp - _restDensity));
            p2 = (1.0f/nPressTemp);
            //printf("p2 %f den %f rest %f gs %f\n",p2,nPartDenTemp,_restDensity,_gasConstant);
            pressWeightTemp = pressureWeighting(curPartPos,nPartPosTemp,_smoothingLength,pressKernConst);
            pressureForce += ( p1 + p2 ) * _particleMass * pressWeightTemp;


            //calculate our viscosity force
            //if the density is zero then we will get NAN's in our devide
            //when density is very small viscosity becomes very unstable so best to have a limiter
            if(nPartDenTemp>0){
                viscWeightTemp = viscosityWeighting(curPartPos,nPartPosTemp,_smoothingLength,viscKernConst);
                viscosityForce += (curPartVel - nParticleData[i].vel) * (_particleMass/nPartDenTemp) * viscWeightTemp;
                //this is needed for surface tension
                massDivDen = _particleMass/nPartDenTemp;
                tensionSum += massDivDen * pressWeightTemp;
                tensionSumTwo += massDivDen * viscWeightTemp;
            }
        }


        //finish our fource calculations
//        pressureForce *= -density;
        //printf("visc: %f,%f,%f\n",viscosityForce.x,viscosityForce.y,viscosityForce.z);
//        viscosityForce *= _visCoef;


//        //calculate our surface tension
//        float nLength = length(tensionSumTwo);
//        float3 tensionForce = make_float3(0);
//        //1.0 is currently our threshold as tension becomes very unstable as n approaches 0
//        if(nLength>0.5){
//            //this also needs to be multipied by our tension contant
//            tensionForce = (tensionSumTwo/nLength) * tensionSum;
//        }



        //calculate our acceleration
        float3 gravity = make_float3(0.0f,-9.8f,0.0f);
//        if(pressureForce.y!=pressureForce.y){
//            printf("pressure force %f,%f,%f density %f\n",pressureForce.x,pressureForce.y,pressureForce.z,density);
//        }
        float3 acc = gravity;
        if(samples>1){
            acc-= pressureForce;
            //acc+= (viscosityForce /*+ tensionForce*/)/density;

        }

        //calculate our new velocity
        //euler intergration (Rubbish over large time steps)
//        float3 newVel = curPartVel + (acc * _timestep);
//        float3 newPos = curPartPos + (newVel * _timestep);

        //leap frog integration
        //more stable if we move by half steps than full
        float3 velHalfBack = curPartVel - 0.5f * _timestep * d_accArray[particleIdx];
        float3 velHalfFor = velHalfBack + _timestep * acc;
        //XSPH velocity correction
        //can be found in Paiva, A., Petronetto, F., Lewiner, T. and Tavares, G. (2009).
        //Particle-based viscoplastic fluid/solid simulation,
        //To achieve this lets take advantage of our shared memory again
//        if(threadIdx.x<samples){
//            nParticleData[threadIdx.x].vel = velHalfFor;
//        }
//        float3 newVel = make_float3(.0f);
//        for(i=0;i<samples;i++){
//            newVel += (2.0f*_particleMass/(density+nParticleData[i].density)) * (nParticleData[i].vel - velHalfBack) * densityWeighting(curPartPos,nParticleData[i].pos,_smoothingLength,densKernConst);
//        }

        float3 newVel = velHalfFor;// + 0.1f * newVel;
        float3 newPos = curPartPos + (newVel * _timestep);

//        if(newPos.x!=newPos.x){
//            printf("shit currentpos %f,%f,%f newPos %f,%f,%f\n pressure force %f,%f,%f samples %d\n",curPartPos.x,curPartPos.y,curPartPos.z,newPos.x,newPos.y,newPos.z,pressureForce.x,pressureForce.y,pressureForce.z,samples);
//            printf("vel: %f,%f,%f\n",newVel.x,newVel.y,newVel.z);
//            printf("timeStep %f acc %f,%f,%f\n\n",_timestep,acc.x,acc.y,acc.z);
//        }
        //update our particle positin and velocity
        d_velArray[particleIdx] = newVel;
        d_posArray[particleIdx] = newPos;
        d_accArray[particleIdx] = acc;

    }
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void fluidSolverKernalDP(float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int *d_cellOccArray, unsigned int *d_cellIndxArray, unsigned int _maxNumThreads,float _smoothingLength, float _timestep, float _particleMass, float _restDensity, float _gasConstant, float _visCoef, float densKernConst, float pressKernConst, float viscKernConst){
    // Read in our how many particles our cell holds
    unsigned int cellOcc = d_cellOccArray[blockIdx.x];
    // Calculate our index for these particles in our buffer
    unsigned int particleIdx = d_cellIndxArray[blockIdx.x];

    // Based on how many particles we have lets calculate how many threads
    // and blocks we need for our kernal launch
    int blocks = 1;
    int threads = cellOcc;
    if(cellOcc>_maxNumThreads){
        blocks = ceil((float)cellOcc/(float)_maxNumThreads)+1;
        threads = _maxNumThreads;
    }

    // Now lets use some dynamic parallism! *Gasps*
    // Lauching a new kernal means we can have as many or as little particles
    // in a cell as we like. However the accuracy of our calculations depends
    // on the ratio of the number of particles per cell and the number of
    // sameples in our SPH calculations. More particles than samples means
    // less accuracy. More samples means more computation
    if(cellOcc>0){
        fluidSolverPerCellKernal<<<blocks,threads>>>(d_posArray,d_velArray,d_accArray,particleIdx,cellOcc,_smoothingLength, _timestep,_particleMass,_restDensity,_gasConstant,_visCoef,densKernConst,pressKernConst,viscKernConst);
    }
}
//----------------------------------------------------------------------------------------------------------------------
__global__ void collisionDetKernal(planeProp *d_planeArray, unsigned int _numPlanes, float3 *d_posArray, float3 *d_velArray, unsigned int _numParticles, float _timeStep){
    //Create our idx
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    //Make sure we're in our range
    if(idx<_numParticles){
        // Exploit some shared memory for fast access of our plane information
        extern __shared__ planeProp planes[];
        if(threadIdx.x<_numPlanes){
            planes[threadIdx.x] = d_planeArray[threadIdx.x];
        }


        // start and end points of our line segement
        float3 vel = d_velArray[idx];
        float3 newVel = vel;
        float3 pos = d_posArray[idx];
        float3 oldVel;
        float t = 0;
        bool intersect = false;
        //iterate through planes
        for(int i=0; i<_numPlanes; i++){
            //if its on the wrong side of the plane move it back and reflect the velocity
            //this is not 100% accurate collision, but at least it works
            if(dot(pos-planes[i].pos,planes[i].normal)<0.0f){
                t = dot(planes[i].pos,planes[i].normal) - dot(planes[i].normal,pos);
                if(length(vel)!=0){
                    t/= dot(planes[i].normal,vel);
                }

                pos = pos + vel * t;
                newVel = newVel - (2.0f * dot(newVel,planes[i].normal) * planes[i].normal);
                oldVel = newVel;
                newVel.x -= (1.0f - planes[i].restCoef) * newVel.x * abs(planes[i].normal.x);
                newVel.y -= (1.0f - planes[i].restCoef) * newVel.y * abs(planes[i].normal.y);
                newVel.z -= (1.0f - planes[i].restCoef) * newVel.z * abs(planes[i].normal.z);
                intersect = true;
            }
        }

        if((pos.x!=pos.x)||(pos.y!=pos.y)||(pos.z!=pos.z)){
            //printf("shit currentpos %f,%f,%f newPos %f,%f,%f vel %f,%f,%f\n",d_posArray[idx].x,d_posArray[idx].y,d_posArray[idx].z,pos.x,pos.y,pos.z,vel.x,vel.y,vel.z);
        }

        //if intersect has occured move our particle back and change our velocity
        if(intersect==true){
            d_posArray[idx] = pos;
            d_velArray[idx] = newVel;
        }
    }
}

//----------------------------------------------------------------------------------------------------------------------
void createHashTable(unsigned int* d_hashArray, float3* d_posArray, unsigned int _numParticles, float _smoothingLength, float _gridSize, int _maxNumThreads){
    //std::cout<<"createHashTable"<<std::endl;
    //calculate how many blocks we want
    int blocks = ceil(_numParticles/_maxNumThreads)+1;
    pointHash<<<_numParticles,1>>>(d_hashArray,d_posArray,_numParticles,_gridSize/_smoothingLength,1.0f/_gridSize);


    //DEBUG: uncomment to print out counted cell occupancy, WARNING SUPER SLOW!
    //thrust::device_ptr<unsigned int> t_hashPtr = thrust::device_pointer_cast(d_hashArray);
    //thrust::copy(t_hashPtr, t_hashPtr+_numParticles, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<"\n"<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("createHashTable CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void sortByKey(unsigned int *d_hashArray, float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int _numParticles){
    //std::cout<<"sortByKey"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //thrusts sort algorithm
    thrust::device_ptr<unsigned int> t_hashPtr = thrust::device_pointer_cast(d_hashArray);
    thrust::device_ptr<float3> t_posPtr = thrust::device_pointer_cast(d_posArray);
    thrust::device_ptr<float3> t_velPtr = thrust::device_pointer_cast(d_velArray);
    thrust::device_ptr<float3> t_accPtr = thrust::device_pointer_cast(d_accArray);


    //sort our buffers
    thrust::sort_by_key(t_hashPtr,t_hashPtr+_numParticles, thrust::make_zip_iterator(thrust::make_tuple(t_posPtr,t_velPtr,t_accPtr)));


    //DEBUG: uncomment to print out sorted hash keys
    //thrust::copy(t_hashPtr, t_hashPtr+_numParticles, std::ostream_iterator<unsigned int>(std::cout, " "));
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("sortByKey CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}
//----------------------------------------------------------------------------------------------------------------------
void countCellOccupancy(unsigned int *d_hashArray, unsigned int *d_cellOccArray,unsigned int _hashTableSize, unsigned int _numPoints, unsigned int _maxNumThreads){
    //std::cout<<"countCellOccupancy"<<std::endl;
    //calculate how many blocks we want
    int blocks = ceil(_numPoints/_maxNumThreads)+1;
    countCellOccKernal<<<blocks,_maxNumThreads>>>(d_hashArray,d_cellOccArray,_hashTableSize,_numPoints);


    //DEBUG: uncomment to print out counted cell occupancy
    //thrust::device_ptr<unsigned int> t_occPtr = thrust::device_pointer_cast(d_cellOccArray);
    //thrust::copy(t_occPtr, t_occPtr+_hashTableSize, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<"\n"<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("countCellOccupancy CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void fillUint(unsigned int *_pointer, unsigned int _arraySize, unsigned int _fill){
    //std::cout<<"fillUint"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //them in thrust fill
    thrust::device_ptr<unsigned int> t_Ptr = thrust::device_pointer_cast(_pointer);
    //fill our buffer
    thrust::fill(t_Ptr, t_Ptr+_arraySize, _fill);

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("FillUint CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }

}
//----------------------------------------------------------------------------------------------------------------------
void createCellIdx(unsigned int* d_cellOccArray, unsigned int _size,unsigned int* d_cellIdxArray){
    //std::cout<<"createCellIdx"<<std::endl;
    //Turn our raw pointers into thrust pointers so we can use
    //them in thrust
    thrust::device_ptr<unsigned int> t_cellOccPtr = thrust::device_pointer_cast(d_cellOccArray);
    thrust::device_ptr<unsigned int> t_cellIdxPtr = thrust::device_pointer_cast(d_cellIdxArray);
    //run an excludive scan on our arrays
    thrust::exclusive_scan(t_cellOccPtr,t_cellOccPtr+_size,t_cellIdxPtr);

    //DEBUG: uncomment to print out cell index buffer
    //thrust::copy(t_cellIdxPtr, t_cellIdxPtr+_size, std::ostream_iterator<unsigned int>(std::cout, " "));
    //std::cout<<std::endl;
    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("createCellIdx CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}
//----------------------------------------------------------------------------------------------------------------------
void fluidSolver(float3 *d_posArray, float3 *d_velArray, float3 *d_accArray, unsigned int *d_cellOccArray, unsigned int *d_cellIndxArray, unsigned int _hashTableSize, unsigned int _maxNumThreads, float _smoothingLength, float _timestep, float _particleMass, float _restDensity, float _gasConstant, float _visCoef, float _densKernConst, float _pressKernConst, float _viscKernConst){
    //std::cout<<"fluidSolver"<<std::endl;
    //printf("memory allocated: %d",_maxNumThreads*(sizeof(particleProp)));
    //fluidSolverKernal<<<_hashTableSize, 30>>>(d_posArray,d_velArray,d_cellOccArray,d_cellIndxArray,_smoothingLength,_timestep, _particleMass, _restDensity,_gasConstant,_visCoef, _densKernConst, _pressKernConst, _viscKernConst);


    fluidSolverKernalDP<<<_hashTableSize, 1>>>(d_posArray,d_velArray,d_accArray,d_cellOccArray,d_cellIndxArray,_maxNumThreads,_smoothingLength,_timestep, _particleMass, _restDensity,_gasConstant,_visCoef, _densKernConst, _pressKernConst, _viscKernConst);
    //std::cout<<std::endl;

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("Fluid solver CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
}

//----------------------------------------------------------------------------------------------------------------------
void collisionDetectionSolver(planeProp *d_planeArray, unsigned int _numPlanes, float3 *d_posArray, float3 *d_velArray, float _timeStep, unsigned int _numParticles, unsigned int _maxNumThreads){
    //calculate how many blocks we want
    int blocks = ceil(_numParticles/_maxNumThreads)+1;
    //launch collision solver
    collisionDetKernal<<<blocks,_maxNumThreads,_numPlanes*sizeof(planeProp)>>>(d_planeArray,_numPlanes,d_posArray,d_velArray,_numParticles,_timeStep);

}
//----------------------------------------------------------------------------------------------------------------------
